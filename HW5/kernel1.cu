#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void mandelKernel(int* d_out, int width, float stepX, float stepY, float lowerX, float lowerY, int count) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;

    int iter;
    for (iter = 0; iter < count; ++iter){
	if (z_re * z_re + z_im * z_im > 4.f) {
        break;
    }

	float new_re = z_re * z_re - z_im * z_im;
	float new_im = 2.f * z_re * z_im;
	z_re = c_re + new_re;
	z_im = c_im + new_im;
    }

    int idx = thisX + thisY * width;
    d_out[idx] = iter;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    //int N = resX * resY;
    int size = resX * resY * sizeof(int);

    // allocate the host memory
    int *data;
    data = (int*) malloc(size);
    // allocate GPU memory
    int *d_out;
    hipMalloc(&d_out, size);

    // CUDA function
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(d_out, resX, stepX, stepY, lowerX, lowerY, maxIterations);

    hipMemcpy(data, d_out, size, hipMemcpyDeviceToHost);
    memcpy(img, data, size);

    // Free allocated memory
    hipFree(d_out);
    free(data);
}